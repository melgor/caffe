#include "hip/hip_runtime.h"
// Copyright 2014 BVLC and contributors.

#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"
#include "caffe/util/math_functions.hpp"

#define max(a, b) ((a < b) ? b : a)
#define min(a, b) ((a < b) ? a : b)

namespace caffe {

template <typename Dtype>
__global__ void MaxPoolForward(const int nthreads, const Dtype* bottom_data,
    const int num, const int channels,
    const int height, const int width,
    const int roi_start_h, const int roi_start_w,
    const int roi_end_h, const int roi_end_w,
    const int bin_num_h, const int bin_num_w,
    const float bin_size_h, const float bin_size_w,
    Dtype* top_data, int* mask, Dtype* top_mask) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int pw = index % bin_num_w;
    int ph = (index / bin_num_w) % bin_num_h;
    int c = (index / bin_num_w / bin_num_h) % channels;
    int n = index / bin_num_w / bin_num_h / channels;
    int hstart = roi_start_h + max(floor(ph * bin_size_h), 0);
    int wstart = roi_start_w + max(floor(pw * bin_size_w), 0);
    int hend = min(roi_start_h + ceil((ph + 1) * bin_size_h), roi_end_h);
    int wend = min(roi_start_w + ceil((pw + 1) * bin_size_w), roi_end_w);
    Dtype maxval = -FLT_MAX;
    int maxidx = -1;
    bottom_data += (n * channels + c) * width * height;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        if (bottom_data[h * width + w] > maxval) {
          maxidx = h * width + w;
          maxval = bottom_data[maxidx];
        }
      }
    }
    top_data[index] = maxval;
    if (mask) {
      mask[index] = maxidx;
    } else {
      top_mask[index] = maxidx;
    }
  }
}

template <typename Dtype>
void PyramidLevelLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  int count = top[0]->count();
  // We'll output the mask to top[1] if it's of size >1.
  const bool use_top_mask = top.size() > 1;
  int* mask = NULL;
  Dtype* top_mask = NULL;
  switch (this->layer_param_.pyramid_level_param().pool()) {
  case PyramidLevelParameter_PoolMethod_MAX:
    if (use_top_mask) {
      top_mask = top[1]->mutable_gpu_data();
    } else {
      mask = max_idx_->mutable_gpu_data();
    }
    // NOLINT_NEXT_LINE(whitespace/operators)
    MaxPoolForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, bottom[0]->num(), channels_,
        height_, width_, roi_start_h_, roi_start_w_, roi_end_h_, roi_end_w_,
        bin_num_h_, bin_num_w_, bin_size_h_, bin_size_w_,
        top_data, mask, top_mask);
    break;
  case PyramidLevelParameter_PoolMethod_AVE:
    NOT_IMPLEMENTED;
    break;
  case PyramidLevelParameter_PoolMethod_STOCHASTIC:
    NOT_IMPLEMENTED;
    break;
  default:
    LOG(FATAL) << "Unknown pooling method.";
  }
  CUDA_POST_KERNEL_CHECK;
}


template <typename Dtype>
__global__ void MaxPoolBackward(const int nthreads, const Dtype* top_diff,
    const int* mask, const Dtype* top_mask, const int num, const int channels,
    const int height, const int width,
    const int roi_start_h, const int roi_start_w,
    const int roi_end_h, const int roi_end_w,
    const int bin_num_h, const int bin_num_w,
    const float bin_size_h, const float bin_size_w,
    Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    int w = index % width;
    int h = (index / width) % height;
    int c = (index / width / height) % channels;
    int n = index / width / height / channels;
    int phstart = max(floor((h - roi_start_h) / bin_size_h - 1), 0);
    int phend = min(ceil((h - roi_start_h + 1) / bin_size_h), bin_num_h);
    int pwstart = max(floor((w - roi_start_w) / bin_size_w - 1), 0);
    int pwend = min(ceil((w - roi_start_w + 1) / bin_size_w), bin_num_w);
    Dtype gradient = 0;
    int offset = (n * channels + c) * bin_num_h * bin_num_w;
    top_diff += offset;
    if (mask) {
      mask += offset;
      for (int ph = phstart; ph < phend; ++ph) {
        for (int pw = pwstart; pw < pwend; ++pw) {
          if (mask[ph * bin_num_w + pw] == h * width + w) {
            gradient += top_diff[ph * bin_num_w + pw];
          }
        }
      }
    } else {
      top_mask += offset;
      for (int ph = phstart; ph < phend; ++ph) {
        for (int pw = pwstart; pw < pwend; ++pw) {
          if (top_mask[ph * bin_num_w + pw] == h * width + w) {
            gradient += top_diff[ph * bin_num_w + pw];
          }
        }
      }
    }
    bottom_diff[index] = gradient;
  }
}

template <typename Dtype>
void PyramidLevelLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  const int count = bottom[0]->count();
  caffe_gpu_set(count, Dtype(0.), bottom_diff);
  // We'll output the mask to top[1] if it's of size >1.
  const bool use_top_mask = top.size() > 1;
  const int* mask = NULL;
  const Dtype* top_mask = NULL;
  switch (this->layer_param_.pyramid_level_param().pool()) {
  case PyramidLevelParameter_PoolMethod_MAX:
    if (use_top_mask) {
      top_mask = top[1]->gpu_data();
    } else {
      mask = max_idx_->gpu_data();
    }
    // NOLINT_NEXT_LINE(whitespace/operators)
    MaxPoolBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, mask, top_mask, top[0]->num(), channels_,
        height_, width_, roi_start_h_, roi_start_w_, roi_end_h_, roi_end_w_,
        bin_num_h_, bin_num_w_, bin_size_h_, bin_size_w_, bottom_diff);
    break;
  case PyramidLevelParameter_PoolMethod_AVE:
    NOT_IMPLEMENTED;
    break;
  case PyramidLevelParameter_PoolMethod_STOCHASTIC:
    NOT_IMPLEMENTED;
    break;
  default:
    LOG(FATAL) << "Unknown pooling method.";
  }
  CUDA_POST_KERNEL_CHECK;
}


INSTANTIATE_LAYER_GPU_FUNCS(PyramidLevelLayer);


}  // namespace caffe